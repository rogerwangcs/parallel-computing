#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "../timerc.h"

/*--------------------------------------------------------------
 *    Lower Triangular Matrix Inverse
 *    CPU RECURSION, GPU MAT MULT
 *    Solves Lower Triangle System of Equations
 *
 *    Author: Roger Wang
 *-------------------------------------------------------------- */

// Helper Functions
void printSubMat(double *a, int idx, int size, int n, const char *title) {
    printf("%s\n", title);
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            int local_idx = idx + i * n + j;
            printf("%5.2f ", a[local_idx]);
        }
        printf("\n");
    }
    printf("\n");
}

void printMat(double *a, int n, const char *title) {
    printf("%s\n", title);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%5.2f ", a[i * n + j]);
        }
        printf("\n");
    }
    printf("\n\n");
}

void printVec(double *b, int n, const char *title) {
    printf("%s\n", title);
    for (int i = 0; i < n; i++) {
        printf("%5.2f ", b[i]);
    }
    printf("\n");
}

double *initVec(int n) {
    double *b = (double *)malloc(n * sizeof(double));
    for (int i = 0; i < n; i++) {
        b[i] = rand() / (double)RAND_MAX;
    }
    return b;
}

double *initMat(int n) {
    double *a = (double *)malloc(n * n * sizeof(double));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i >= j)
                a[i * n + j] = rand() / (double)RAND_MAX;
            else
                a[i * n + j] = 0;
        }
    }
    return a;
}

__host__ __device__ void copyMat(double *newA, double *a, int idx, int size, int n) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            int local_idx = idx + i * n + j;
            a[local_idx] = newA[i * size + j];
        }
    }
}

void copyVec(double *newB, double *b, int n) {
    for (int i = 0; i < n; i++) {
        newB[i] = b[i];
    }
}

// device kernel function for multMat parallel
__global__ void multMat_kernel(double *a_d, int idx_1, int idx_2, int size, int n, double *res) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    res[i * size + j] = 0.0;
    for (int k = 0; k < size; k++) {
        printf("%d ", k);
        res[i * size + j] += a_d[idx_1 + i * n + k] * a_d[idx_2 + k * n + j];
    }
    return;
}

// Parallel version
void multMat(double *a_d, int idx_1, int idx_2, int size, int n, double *res_d) {
    multMat_kernel<<<size, size>>>(a_d, idx_1, idx_2, size, n, res_d);
    hipDeviceSynchronize();
    return;
}

void multMatVec(double *a, double *b, double *x, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            x[i] += a[i * n + j] * b[j];
        }
    }
}

void checkInverse(double *a, double *a_inv, double *res, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            res[i * n + j] = 0.0;
            for (int k = 0; k < n; k++)
                res[i * n + j] += a[i * n + k] * a_inv[k * n + j];
        }
    }
}

void addNegative(double *a, int idx, int size, int n) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            int local_idx = idx + i * n + j;
            a[local_idx] = -1 * a[local_idx];
        }
    }
}

// Base Case 2x2 Inplace inversion
__host__ __device__ void invertTwoByTwo(double *a, int idx, int n) {
    int idx0 = idx;
    int idx1 = idx + 1;
    int idx2 = idx + n;
    int idx3 = idx + n + 1;
    double det = a[idx0] * a[idx3] - a[idx2] * a[idx1];
    double temp;
    temp = a[idx3];
    a[idx3] = a[idx0];
    a[idx0] = temp;
    a[idx2] = -a[idx2];
    a[idx1] = -a[idx1];

    a[idx0] *= (1 / det);
    a[idx1] *= (1 / det);
    a[idx2] *= (1 / det);
    a[idx3] *= (1 / det);
}

// Recursively invert A11, A22, and then A21
void inverseRecurseParallelMult(double *a_d, int idx, int size, int n) {
    // Base Case: Invert if a is a simple 2x2 matrix
    if (size == 2) {
        invertTwoByTwo(a_d, idx, n);
        return;
    }

    // Calculate starting index for the 3 submatrices and make recursive calls
    int a11_idx = idx;                                      // upper left submatrix
    int a22_idx = idx + size / 2 * n + size / 2;            // lower right submatrix
    int a21_idx = idx + size / 2 * n;                       // bottom left full submatrix
    inverseRecurseParallelMult(a_d, idx, size / 2, n);      // recurse on upper left submatrix
    inverseRecurseParallelMult(a_d, a22_idx, size / 2, n);  // recurse on bottom right submatrix

    // Invert A21
    printSubMat(a_d, a22_idx, size / 2, n, "A22");
    printSubMat(a_d, a21_idx, size / 2, n, "A21");
    printSubMat(a_d, a11_idx, size / 2, n, "A11");

    double *res_d;
    hipMalloc((void **)&res_d, size * size * sizeof(double));

    multMat(a_d, a22_idx, a21_idx, size / 2, n, res_d);  // A22 * A21 // A22 * A21
    // printMat(res, size / 2, "A22 * A21");
    copyMat(res_d, a_d, a21_idx, size / 2, n);           // Put result into A21
    multMat(a_d, a22_idx, a21_idx, size / 2, n, res_d);  // A22 * A21  // A21 * A11
    // printMat(res, size / 2, "result");
    copyMat(res_d, a_d, a21_idx, size / 2, n);  // Put result into A21
    addNegative(a_d, a21_idx, size / 2, n);     // Add negative sign to A21
                                                // printSubMat(a_d, a21_idx, size / 2, n, "result");
    hipFree(res_d);

    return;
}

// sets up and calls inverseRecurse
void inverseRecurseHelper(double *a, int n) {
    // device variables
    double *a_d;
    hipMalloc((void **)&a_d, n * n * sizeof(double));
    hipMemcpy(a_d, a, n * n * sizeof(double), hipMemcpyHostToDevice);
    printf("nice");
    inverseRecurseParallelMult(a_d, 0, n, n);

    // copy and return calculations
    hipMemcpy(a, a_d, n * n * sizeof(double), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(a_d);
    return;
}

int main() {
    printf("nice");
    float cpu_time;
    float gpu_time;
    int n = pow(2, 2);  // Matrix size: 2^x = n
    double *a, *a_old, *b, *b_old;
    double *x = (double *)malloc(n * sizeof(double));
    a = initMat(n);
    a_old = initMat(n);
    copyMat(a_old, a, 0, n, n);
    b = initVec(n);
    b_old = initVec(n);
    copyVec(b_old, b, n);

    // printMat(a, n, "Initial Matrix:");
    cstart();
    gstart();
    inverseRecurseHelper(a, n);
    gend(&gpu_time);
    cend(&cpu_time);
    printf("GPU matrix inverse time: %f\n", cpu_time + gpu_time);
    printMat(a, n, "Inverted Matrix:");

    // Double check matrix inversion
    double *res = (double *)malloc(n * n * sizeof(double));
    checkInverse(a, a_old, res, n);
    printMat(res, n, "A * A^-1 (should be identity mat):");
    free(res);

    // Get Solution
    multMatVec(a, b, x, n);
    printVec(b, n, "x: (Solution)");
    // Check Solution
    multMatVec(a, x, b, n);
    printVec(b, n, "b:");
    // printVec(b_old, n, "test_b: (should equal b)");

    free(x);
    free(a);
    free(a_old);
    free(b);
    free(b_old);
    return 0;
}