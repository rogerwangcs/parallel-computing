#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "check_solution.h"
#include "helpers.h"
#include "timerc.h"

/*--------------------------------------------------------------
 *    Lower Triangular Matrix Inverse
 *    GPU Iterative, CPU MAT MULT
 *    Run: Driver ( with -arch=sm_35 -rdc=true)
 *    Author: Roger Wang
 *-------------------------------------------------------------- */

__device__ void invertHelperParallelIter(double *a_d, int idx, int size, int n) {
    // Base Case: Invert if a is a simple 2x2 matrix
    if (size == 2) {
        invertTwoByTwo(a_d, idx, n);
        return;
    }

    // Calculate starting index for the 3 submatrices
    int a11_idx = idx;                            // upper left submatrix
    int a22_idx = idx + size / 2 * n + size / 2;  // lower right submatrix
    int a21_idx = idx + size / 2 * n;             // bottom left full submatrix

    // Invert A21
    double *res_d;
    hipMalloc((void **)&res_d, size * size * sizeof(double));

    multSubMat(a_d, a22_idx, a21_idx, size / 2, n, res_d);  // A22 * A21
    copyMat(res_d, a_d, a21_idx, size / 2, n);              // Put result into A21
    multSubMat(a_d, a21_idx, a11_idx, size / 2, n, res_d);  // A21 * A11
    copyMat(res_d, a_d, a21_idx, size / 2, n);              // Put result into A21
    addNegative(a_d, a21_idx, size / 2, n);                 // Add negative sign to A21
    hipFree(res_d);
}

__global__ void invertPass_kernel(double *a_d, int total_ops, int n) {
    int block_size = n / total_ops;
    int inv_idx = threadIdx.x * ((n * n / total_ops) + block_size);
    // printf("Inv_idx: %d Block Size: %d\n", inv_idx, block_size);
    invertHelperParallelIter(a_d, inv_idx, block_size, n);
}

void invertBottomUpParallelIter(double *a_d, int n) {
    int total_iter = log2(n);
    for (int iter = 0; iter < total_iter; iter++) {
        int total_ops = pow(2, total_iter - iter - 1);
        invertPass_kernel<<<1, total_ops>>>(a_d, total_ops, n);
        hipDeviceSynchronize();
    }
}

int gpu_iterative_cpu_mult(int inputSize, int check, int debug) {
    printf("\nGPU Iterative, CPU Multiplication\n");
    // Initialize
    int n = pow(2, inputSize);  // Matrix size: 2^x = n
    double *a, *a_old, *b, *b_old;
    a = initMat(n);
    a_old = initMat(n);
    copyMat(a_old, a, 0, n, n);
    b = initVec(n);
    b_old = initVec(n);
    copyVec(b_old, b, n);

    // Start Inversion
    float cpu_time, cpu_time2;
    float gpu_time;
    cstart();
    double *a_d;
    hipMalloc((void **)&a_d, n * n * sizeof(double));
    hipMemcpy(a_d, a, n * n * sizeof(double), hipMemcpyHostToDevice);
    cend(&cpu_time);
    gstart();
    invertBottomUpParallelIter(a_d, n);
    gend(&gpu_time);
    cstart();
    hipMemcpy(a, a_d, n * n * sizeof(double), hipMemcpyDeviceToHost);
    cend(&cpu_time2);
    hipFree(a_d);
    printf("Total Time: %f\n", cpu_time + cpu_time2 + gpu_time);
    // End Inversion

    // Check Solution
    if (check)
        checkSolution(a, a_old, b_old, n, debug);

    free(a);
    free(a_old);
    free(b);
    free(b_old);
    return 0;
}
