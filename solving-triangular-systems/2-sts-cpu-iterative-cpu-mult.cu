#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "../timerc.h"

/*--------------------------------------------------------------
 *    Lower Triangular Matrix Inverse
 *    CPU Iterative, CPU MAT MULT
 *    Solves Lower Triangle System of Equations

 *    Author: Roger Wang
 *-------------------------------------------------------------- */

void invertHelper(double *a, int idx, int size, int n) {
    // Base Case: Invert if a is a simple 2x2 matrix
    if (size == 2) {
        invertTwoByTwo(a, idx, n);
        return;
    }

    // Calculate starting index for the 3 submatrices
    int a11_idx = idx;                            // upper left submatrix
    int a22_idx = idx + size / 2 * n + size / 2;  // lower right submatrix
    int a21_idx = idx + size / 2 * n;             // bottom left full submatrix

    // Invert A21
    double *res = (double *)malloc(size * size * sizeof(double));
    multSubMat(a, a22_idx, a21_idx, size / 2, n, res);  // A22 * A21
    copyMat(res, a, a21_idx, size / 2, n);              // Put result into A21
    multSubMat(a, a21_idx, a11_idx, size / 2, n, res);  // A21 * A11
    copyMat(res, a, a21_idx, size / 2, n);              // Put result into A21
    addNegative(a, a21_idx, size / 2, n);               // Add negative sign to A21
    free(res);
}

void invertBottomUp(double *a, int n) {
    int total_iter = log2(n);
    for (int iter = 0; iter < total_iter; iter++) {
        int total_ops = pow(2, total_iter - iter - 1);
        for (int op_idx = 0; op_idx < total_ops; op_idx++) {
            int block_size = n / total_ops;
            int inv_idx = op_idx * ((n * n / total_ops) + block_size);
            // printf("Inv_idx: %d Block Size: %d\n", inv_idx, block_size);
            invertHelper(a, inv_idx, block_size, n);
        }
    }
}

// Just provide array input size (power of 2)
int cpu_iterative_cpu_mult(int inputSize, int check, int debug) {
    printf("\nCPU Iterative, CPU Multiplication\n");
    // Initialize
    float cpu_time;
    int n = pow(2, inputSize);  // Matrix size: 2^x = n
    double *a, *a_old, *b, *b_old;
    a = initMat(n);
    a_old = initMat(n);
    copyMat(a_old, a, 0, n, n);
    b = initVec(n);
    b_old = initVec(n);
    copyVec(b_old, b, n);

    // Begin Inversion
    cstart();
    invertBottomUp(a, n);
    cend(&cpu_time);
    printf("Total Time: %f\n", cpu_time);
    // End Inversion

    // Check Solution
    if (check)
        checkSolution(a, a_old, b_old, n, debug);

    free(a);
    free(a_old);
    free(b);
    free(b_old);
    return 0;
}