#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "../timerc.h"

/*--------------------------------------------------------------
 *    Lower Triangular Matrix Inverse On GPU
 *    Returns the inverse of a lower triangular matrix recursively
 *
 *    Author: Roger Wang
 *-------------------------------------------------------------- */

// Helper Functions
void printSubMat(double *a, int idx, int size, int n, const char *title) {
    printf("%s\n", title);
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            int local_idx = idx + i * n + j;
            printf("%5.2f ", a[local_idx]);
        }
        printf("\n");
    }
    printf("\n");
}

void printMat(double *a, int n, const char *title) {
    printf("%s\n", title);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%5.2f ", a[i * n + j]);
        }
        printf("\n");
    }
    printf("\n\n");
}

void printVec(double *b, int n, const char *title) {
    printf("%s\n", title);
    for (int i = 0; i < n; i++) {
        printf("%5.2f ", b[i]);
    }
    printf("\n");
}

double *initMat(int n) {
    double *a = (double *)malloc(n * n * sizeof(double));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i >= j)
                a[i * n + j] = i * n + j + 1;
            else
                a[i * n + j] = 0;
        }
    }
    return a;
}

double *initVec(int n) {
    double *b = (double *)malloc(n * sizeof(double));
    for (int i = 0; i < n; i++) {
        b[i] = i;
    }
    return b;
}

void copyMat(double *newA, double *a, int idx, int size, int n) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            int local_idx = idx + i * n + j;
            a[local_idx] = newA[i * size + j];
        }
    }
}

// device kernel function for multMat parallel
__global__ void multMat_kernel(double *a, int idx_1, int idx_2, int size, int n, double *res) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    res[i * size + j] = 0.0;
    for (int k = 0; k < size; k++) {
        res[i * size + j] += a[idx_1 + i * n + k] * a[idx_2 + k * n + j];
    }
    return;
}

// Parallel version
void multMat(double *a, int idx_1, int idx_2, int size, int n, double *res) {
    // device variables
    double *a_d;
    double *res_d;
    hipMalloc((void **)&a_d, size * size * sizeof(double));
    hipMalloc((void **)&res_d, size * size * sizeof(double));
    hipMemcpy(a_d, a, size * size * sizeof(double), hipMemcpyHostToDevice);

    // execute kernel
    multMat_kernel<<<size, size>>>(a_d, idx_1, idx_2, size, n, res_d);
    hipDeviceSynchronize();

    // copy and return calculations
    hipMemcpy(res, res_d, size * size * sizeof(double), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(a_d);
    hipFree(res_d);
    return;
}

void multMatVec(double *a, double *b, double *x, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            x[i] += a[i * n + j] * b[j];
        }
    }
}

void checkInverse(double *a, double *a_inv, double *res, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            res[i * n + j] = 0.0;
            for (int k = 0; k < n; k++)
                res[i * n + j] += a[i * n + k] * a_inv[k * n + j];
        }
    }
}

void addNegative(double *a, int idx, int size, int n) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            int local_idx = idx + i * n + j;
            a[local_idx] = -1 * a[local_idx];
        }
    }
}

// Base Case 2x2 Inplace inversion
__host__ __device__ void invertTwoByTwo(double *a, int idx, int n) {
    int idx0 = idx;
    int idx1 = idx + 1;
    int idx2 = idx + n;
    int idx3 = idx + n + 1;
    double det = a[idx0] * a[idx3] - a[idx2] * a[idx1];
    double temp;
    temp = a[idx3];
    a[idx3] = a[idx0];
    a[idx0] = temp;
    a[idx2] = -a[idx2];
    a[idx1] = -a[idx1];

    a[idx0] *= (1 / det);
    a[idx1] *= (1 / det);
    a[idx2] *= (1 / det);
    a[idx3] *= (1 / det);
}

// Recursively invert A11, A22, and then A21
void inverseRecurse(double *a, int idx, int size, int n) {
    // Base Case: Invert if a is a simple 2x2 matrix
    if (size == 2) {
        invertTwoByTwo(a, idx, n);
        return;
    }

    // Calculate starting index for the 3 submatrices and make recursive calls
    int a11_idx = idx;                            // upper left submatrix
    int a22_idx = idx + size / 2 * n + size / 2;  // lower right submatrix
    int a21_idx = idx + size / 2 * n;             // bottom left full submatrix
    inverseRecurse(a, idx, size / 2, n);          // recurse on upper left submatrix
    inverseRecurse(a, a22_idx, size / 2, n);      // recurse on bottom right submatrix

    // Invert A21

    double *res = (double *)malloc(size * size * sizeof(double));
    multMat(a, a22_idx, a21_idx, size / 2, n, res);  // A22 * A21
    copyMat(res, a, a21_idx, size / 2, n);           // Put result into A21
    multMat(a, a21_idx, a11_idx, size / 2, n, res);  // A21 * A11
    copyMat(res, a, a21_idx, size / 2, n);           // Put result into A21
    addNegative(a, a21_idx, size / 2, n);            // Add negative sign to A21
    free(res);

    return;
}

int main() {
    float cpu_time;
    int n = pow(2, 10);  // Matrix size: 2^x = n
    double *a, *a_old, *b, *b_old;
    double *x = (double *)malloc(n * sizeof(double));
    a = initMat(n);
    a_old = initMat(n);
    b = initVec(n);
    b_old = initVec(n);

    // printMat(a, n, "Initial Matrix:");
    cstart();
    inverseRecurse(a, 0, n, n);
    cend(&cpu_time);
    printf("GPU matrix inverse time: %f\n", cpu_time);
    // printMat(a, n, "Inverted Matrix:");

    // Double check matrix inversion
    double *res = (double *)malloc(n * n * sizeof(double));
    checkInverse(a, a_old, res, n);
    // printMat(res, n, "A * A^-1 (should be identity mat):");
    free(res);

    // Get Solution
    multMatVec(a, b, x, n);
    // printVec(b, n, "x: (Solution)");
    // Check Solution
    multMatVec(a, x, b, n);
    // printVec(b, n, "b:");
    // printVec(b_old, n, "test_b: (should equal b)");

    free(x);
    free(a);
    free(a_old);
    free(b);
    free(b_old);
    return 0;
}
